#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <random>
#include <fstream>
#include <string>


using namespace thrust;
using namespace std;

#define NUM_OPERATIONS 20
#define NUM_TRANSACTIONS 4000000
#define NUM_RECORDS 1000000
typedef int transactions[NUM_OPERATIONS];

//1, 5, 10, 50, 100, 250, 500, 1000, 2500, 5000, 10000
const int THREADS = 5000;
const int BLOCKS = 10;
const int STRIDE = NUM_TRANSACTIONS / THREADS;
const int NUM_RUNS = 10;

__device__ int lock(int *lock) {
    return atomicCAS(lock, 0, 1);
}

__device__ void unlock(int *lock) {
    atomicExch(lock, 0);
}

__device__ void insertion_sort(int *arr)
{
    int i, key, j;
    for (i = 1; i < NUM_OPERATIONS; i++) {
        key = arr[i];
        j = i - 1;

        while (j >= 0 && arr[j] > key) {
            arr[j + 1] = arr[j];
            j = j - 1;
        }
        arr[j + 1] = key;
    }
}

__global__ void transaction_worker(int *lock_table, int *table, transactions *d_transactions, int stride) {

    int tid =  blockIdx.x *blockDim.x + threadIdx.x;
    int start = tid * stride;
    int end = (start + stride) - 1;

//    sort phase
    for(int i = start; i < end; ++i) {
        insertion_sort(d_transactions[i]);
    }
    //process transactions
    for(int i = start; i < end; ++i) {
        //growing phase
        int prev = -1;
        int abort_index[NUM_OPERATIONS];
        memset(abort_index, -1, sizeof(abort_index));
        int is_abort = 0;

        for(int j = 0; j < NUM_OPERATIONS; ++j) {
            int idx = d_transactions[i][j];
            if(idx != prev) {
                int is_locked = lock(&lock_table[idx]);
                if(is_locked == 0) {
                    abort_index[j] = idx;
                } else {
                    is_abort = 1;
                    break;
                }
            }
            prev = idx;
        }

        if(is_abort == 0) {
            //critical section
            prev = -1;
            for(int j = 0; j < NUM_OPERATIONS; ++j) {
                int idx = d_transactions[i][j];
                if(idx != prev) {
                    table[idx] += 1;
                }
                prev = idx;
            }

            //shrinking phase
            prev = -1;
            for(int j = 0; j < NUM_OPERATIONS; ++j) {
                int idx = d_transactions[i][j];
                if(idx != prev) {
                    unlock(&lock_table[idx]);
                }
                prev = idx;
            }
        } else {
            for(int j = 0; j < NUM_OPERATIONS; ++j) {
                int idx = abort_index[j];
                if(idx != -1) {
                    unlock(&lock_table[idx]);
                }
                prev = idx;
            }
        }
    }
}

int main() {
    string OUTPUT_FP = "/home/nicoroble/cuda-s2pl/data/cuda-transactions_readsetsize-" + to_string(NUM_OPERATIONS) + "_transactions-" + to_string(NUM_TRANSACTIONS) + "_tablesize-" + to_string(NUM_RECORDS) + ".tsv";

    vector<float> runtimes;

    for(int n = 0; n < NUM_RUNS; ++n) {
        transactions *h_transactions;
        transactions *d_transactions;
        int *lock_table;
        int *table;
        int *d_table;

        size_t dsize = NUM_TRANSACTIONS * NUM_OPERATIONS * sizeof(int);
        hipMalloc((void**)&lock_table, NUM_RECORDS * sizeof(int));
        hipMemset(lock_table, 0, NUM_RECORDS * sizeof(int));

        table = (int*)malloc(sizeof(int) * NUM_RECORDS);
        for(int i =0; i < NUM_RECORDS; ++i) {
            table[i] = 0;
        }


        hipMalloc((void**)&d_table, sizeof(int) * NUM_RECORDS);
        hipMemcpy(d_table, table, sizeof(int) * NUM_RECORDS, hipMemcpyHostToDevice);

        //set memory for host arrays
        h_transactions = (transactions *)malloc(dsize);
        // populate h_ arrays
        memset(h_transactions, 0, dsize);

        random_device rd;
        mt19937 eng(rd());
        uniform_int_distribution<> distribution(0, NUM_RECORDS - 1);

        for(int i = 0; i < NUM_TRANSACTIONS; ++i) {
            for(int j =0; j < NUM_OPERATIONS; ++j) {
                h_transactions[i][j] = distribution(eng);
            }
        }

        // Allocate memory on device
        hipMalloc(&d_transactions, dsize);

        // Do memcopies to GPU
        hipMemcpy(d_transactions, h_transactions, dsize, hipMemcpyHostToDevice);

        float elapsed=0;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);

        transaction_worker<<<THREADS / BLOCKS, BLOCKS>>>(lock_table, d_table, d_transactions, STRIDE);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);

        runtimes.push_back(elapsed);

        free(h_transactions);
        hipFree(d_transactions);
        hipFree(d_table);
        hipFree(lock_table);
        hipDeviceSynchronize();
    }

    ofstream outfile;
    outfile.open(OUTPUT_FP, ios_base::app);
    float average_runtime = accumulate(runtimes.begin(), runtimes.end(), 0.0) / runtimes.size();
    outfile << THREADS << "\t" << average_runtime << "\t" << runtimes.size() << "\n";
    outfile.close();

    return 0;
}